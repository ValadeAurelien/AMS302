#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <ctime>
#include <vector>
#include <hiprand.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>

#define MIN(a, b) (((a)<(b)?(a):(b)))


__device__
float source(float rand_b) { return rand_b; }

__device__
float f_gene(float mu, float sigma, float rand_a, float rand_b)
{
    return -mu * logf( rand_a ) / sigma + source(rand_b);
}

__global__
void trajs(float mu, float sigma, float* parts, unsigned nb_parts, float* rands_a, float* rands_b )
{
    int x = threadIdx.x+blockIdx.x*blockDim.x;

    if (x>=nb_parts) return;
    
    parts[x] = f_gene(mu, sigma, rands_a[x], rands_b[x]);
}

__global__
void make_distrib(float* parts,
		  unsigned nb_parts,
		  unsigned* distrib,
		  unsigned nb_segs,
		  unsigned* below,
		  unsigned* above,
		  float min,
		  float max,
		  unsigned nb_threads)
{
    unsigned x = threadIdx.x+blockIdx.x*blockDim.x;

    if (x>=nb_threads) return;

    unsigned range_size = floorf((float) nb_parts/nb_threads),
	i = x*range_size;
    int seg = floorf( (float) (parts[i]-min)/(max-min)*nb_segs );
    for (i++; i<(x+1)*range_size; i++){
	if ( floorf( (float) (parts[i]-min)/(max-min)*nb_segs ) > seg )
	    seg = (int) floorf( (float) (parts[i]-min)/(max-min)*nb_segs );
	if ( seg<0 ) *below++;
	else if ( seg>nb_segs ) *above++;
	else distrib[ seg ]++;
    }
}	

int main(int argc, char **argv)
{
    if (argc!=5) return -1;
    float mu = atof(argv[1]),
	sigma = atof(argv[2]);
    unsigned nb_parts = atoi(argv[3]),
	nb_segs = atoi(argv[4]);
    float* parts,
	*rands_a,
	*rands_b;
    hipMalloc(&parts, sizeof(float)*nb_parts);
    hipMalloc(&rands_a, sizeof(float)*nb_parts);
    hipMalloc(&rands_b, sizeof(float)*nb_parts);
    dim3 blockSize(512),
	gridSize(ceil((float) nb_parts/512));
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL));
    hiprandGenerateUniform(gen, rands_a, nb_parts);
    hiprandGenerateUniform(gen, rands_b, nb_parts);
    trajs<<<gridSize, blockSize>>>(mu, sigma, parts, nb_parts, rands_a, rands_b);
    thrust::sort(thrust::device, parts, parts+nb_parts);
    unsigned* distrib,
	*above,
	*below;
    hipMalloc(&distrib, sizeof(unsigned)*nb_segs);
    hipMalloc(&below, sizeof(unsigned));
    hipMalloc(&above, sizeof(unsigned));
    make_distrib<<<gridSize, blockSize>>>(parts,
					  nb_parts,
					  distrib,
					  nb_segs,
					  below,
					  above,
					  0, 1,
					  MIN(nb_segs/2, nb_parts/2));
    std::vector<unsigned> h_distrib (nb_segs);
    hipMemcpy(h_distrib.data(), distrib, sizeof(unsigned)*nb_segs, hipMemcpyDeviceToHost);
    // for (int i=0; i<nb_segs; i++)
    //     std::cout << (float) i/nb_segs << " " << h_distrib.at(i) << std::endl;
    return 0;
}